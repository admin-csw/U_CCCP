#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE 5

__global__ void arrayAdd(const int* a, const int* b, int* c, int size) {
    int i = threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int a[SIZE] = {1, 2, 3, 4, 5};
    int b[SIZE] = {10, 20, 30, 40, 50};
    int c[SIZE];

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, SIZE * sizeof(int));
    hipMalloc((void**)&d_b, SIZE * sizeof(int));
    hipMalloc((void**)&d_c, SIZE * sizeof(int));

    hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);

    arrayAdd<<<1, SIZE>>>(d_a, d_b, d_c, SIZE);

    hipMemcpy(c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

    printf("Resultant array (CUDA):\n");
    printf("{1, 2, 3, 4, 5} + {10, 20, 30, 40, 50} = {%d, %d, %d, %d, %d}\n", c[0], c[1], c[2], c[3], c[4]);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
