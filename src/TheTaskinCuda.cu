#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include ""
#include "TheEmployeesSalary.h"

hipError_t thehelperfunc(const double* tarray, double* tnewSalaries, int* tSIZE, int* threadsPerBlock, int* blocksPerGrid);

__global__ void TaskDoer(const double* array, double* newSalaries, int* SIZE) {
    int ID = blockIdx.x * blockDim.x + threadIdx.x;
    if (ID < *SIZE) {
        newSalaries[ID] = array[ID] + (array[ID] * 15/100) + 5000;
    }
}

int main() {
    int size = sizeof(TheArrayOfSalaries) / sizeof(TheArrayOfSalaries[0]);

    double newSalaries[size];

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    hipError_t cudaStatus = thehelperfunc(TheArrayOfSalaries, newSalaries, &size, &threadsPerBlock, &blocksPerGrid);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "The Helper function failed!");
        return 1;
    }


    for (int i = 0; i < size; i++) {
        printf("Employee %d: New Salary = %.2f\n", i + 1, newSalaries[i]);
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Cuda Device Reset failed!");
        return 1;
    }

    return 0;
}

hipError_t thehelperfunc(const double* tarray, double* tnewSalaries, int* tSIZE, int* threadsPerBlock, int* blocksPerGrid) {
    double* deviceArray = 0;
    double* deviceNewSalaries = 0;
    int* deviceSize = 0;

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Cuda Set Device failed!  Do you have a CUDA-capable GPU installed?");
        goto AnError;
    }

    cudaStatus = hipMalloc((void**)&deviceArray, sizeof(double) * (*tSIZE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Cuda Malloc failed!");
        goto AnError;
    }

    cudaStatus = hipMalloc((void**)&deviceNewSalaries, sizeof(double) * *tSIZE);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Cuda Malloc failed!");
        goto AnError;
    }

    cudaStatus = hipMalloc((void**)&deviceSize, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Cuda Malloc failed!");
        goto AnError;
    }

    cudaStatus = hipMemcpy(deviceArray, TheArrayOfSalaries, sizeof(double) * *tSIZE, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Cuda Memcpy failed!");
        goto AnError;
    }

    cudaStatus = hipMemcpy(deviceSize, tSIZE, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Cuda Memcpy failed!");
        goto AnError;
    }

    cudaStatus = hipMemcpy(deviceNewSalaries, tnewSalaries, sizeof(double) * *tSIZE, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Cuda Memcpy failed!");
        goto AnError;
    }

    TaskDoer<<<*blocksPerGrid, *threadsPerBlock>>>(deviceArray, deviceNewSalaries, deviceSize);

    cudaStatus = hipGetLastError();

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "TaskDoer Kernel failed!");
        goto AnError;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Cuda Device Synchronize failed!");
        goto AnError;
    }

    cudaStatus = hipMemcpy(tnewSalaries, deviceNewSalaries, sizeof(double) * *tSIZE, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Cuda Memcpy failed!");
        goto AnError;
    }

AnError:
    hipFree(deviceArray);
    hipFree(deviceNewSalaries);
    hipFree(deviceSize);

    return cudaStatus;
}