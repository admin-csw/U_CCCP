#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("Number of CUDA devices: %d\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device %d: %s\n", i, prop.name);
    }

    return 0;
}
