
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>

int main() {
    const int size = 20;

    // Create a random number geneator
    hiprandGenerator_t generator;
    hiprandStatus_t status = hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);

    if (status != HIPRAND_STATUS_SUCCESS) {
        fprintf(stderr, "Failed to create cuRAND generator\n");
        return 1;
    }

    // Set the seed for the random number generator
    hiprandSetPseudoRandomGeneratorSeed(generator, 1234);

    if (status != HIPRAND_STATUS_SUCCESS) {
        fprintf(stderr, "Failed to set seed for cuRAND generator\n");
        hiprandDestroyGenerator(generator);
        return 1;
    }

    // Allocate memory for the array on the host
    unsigned int* hostArray = new unsigned int[size];

    // Generate random numbers on the device
    status = hiprandGenerate(generator, hostArray, size);
    if (status != HIPRAND_STATUS_SUCCESS) {
        printf("Failed to generate random numbers\n");
        delete[] hostArray;
        hiprandDestroyGenerator(generator);
        return 1;
    }

    // Print the generated random numbers
    for (int i = 0; i < size; i++) {
        printf("%u\n", hostArray[i]);
    }

    printf("\n");

    // Clean up
    hiprandDestroyGenerator(generator);
    delete[] hostArray;

    return 0;
}