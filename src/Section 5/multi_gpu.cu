#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for x + y operation
__global__ void addKernel(int x, int y, int* result) {
    *result = x + y;
}

// CUDA kernel for x * y operation
__global__ void multiplyKernel(int x, int y, int* result) {
    *result = x * y;
}

int main() {
    int x = 5;
    int y = 3;
    int a = 2;
    int b = 4;

    int result_add, result_multiply;

    // GPU device 0
    hipSetDevice(0);

    int *d_result_add;
    hipMalloc((void**)&d_result_add, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL("x"), &x, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL("y"), &y, sizeof(int));

    addKernel<<<1, 1>>>(x, y, d_result_add);
    hipMemcpy(&result_add, d_result_add, sizeof(int), hipMemcpyDeviceToHost);

    // GPU device 1
    hipSetDevice(1);

    int *d_result_multiply;
    hipMalloc((void**)&d_result_multiply, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL("a"), &a, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL("b"), &b, sizeof(int));

    multiplyKernel<<<1, 1>>>(a, b, d_result_multiply);

    hipMemcpy(&result_multiply, d_result_multiply, sizeof(int), hipMemcpyDeviceToHost);

    printf("Result of x + y on GPU 0: %d\n", result_add);
    printf("Result of a * b on GPU 1: %d\n", result_multiply);  

    hipFree(d_result_add);
    hipFree(d_result_multiply);
    return 0;
}
