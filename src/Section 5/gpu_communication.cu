#include <stdio.h>
#include <hip/hip_runtime.h>

#define NUM_GPUS 2

// kernel ufnction to perform computatio on GPU 0
__global__ void kernelA(int *x) {
    *x = (*x) * (*x);
}

// kernel function to perform computation on GPU 1
__global__ void kernelB(int *y) {
    *y = (*y) * 3;
}


int main() {
    int h_data_0; // Host data for GPU 0
    int h_data_1; // Host data for GPU 1
    int *d_data_0; // Device data for GPU 0
    int *d_data_1; // Device data for GPU 1

    hipStream_t stream0, stream1;

    hipSetDevice(0);
    hipStreamCreate(&stream0);
    hipMalloc(&d_data_0, sizeof(int));

    hipSetDevice(1);
    hipStreamCreate(&stream1);
    hipMalloc(&d_data_1, sizeof(int));

    // Enable peer access between GPUs
    hipSetDevice(0);
    hipDeviceEnablePeerAccess(1, 0);

    //  Initialize data on GPU 0
    h_data_0 = 2;
    hipMemcpy(d_data_0, &h_data_0, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel on GPU 0
    kernelA<<<1, 1, 0, stream0>>>(d_data_0);

    // Copy result from GPU 0 to GPU 1
    hipSetDevice(1);
    hipMemcpyPeerAsync(d_data_1, 1, d_data_0, 0, sizeof(int));

    // Launch kernel on GPU 1
    kernelB<<<1, 1, 0, stream1>>>(d_data_1);

    // Copy result back to host from GPU 1 to CPU
    hipMemcpy(&h_data_1, d_data_1, sizeof(int), hipMemcpyDeviceToHost);

    printf("Result: %d\n", h_data_1); // Should print 12 (2^2 * 3 = 12)

    // Free device memory
    hipSetDevice(0);
    hipFree(d_data_0);
    hipSetDevice(1);
    hipFree(d_data_1);

    // Destroy streams
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);

    return 0;
}