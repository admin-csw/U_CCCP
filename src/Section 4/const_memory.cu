#include <stdio.h>
#include <hip/hip_runtime.h>


__constant__ int constData;

__global__ void myKernel() {
    int data = constData;   
    printf("Const Data: %d\n", constData);
}

int main() {
     int hostData = 100;
    hipMemcpyToSymbol(HIP_SYMBOL(constData), &hostData, sizeof(int));

    myKernel<<<1, 1>>>();
    hipDeviceSynchronize();

    return 0;
}