#include <stdio.h>
#include <hip/hip_runtime.h>

#include <iostream>

__global__ void stage1Kernel(int* input, int* intermediate, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        intermediate[tid] += input[tid] + 1;
    }
}

__global__ void stage2Kernel(int* intermediate, int* output, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        output[tid] += intermediate[tid] * 2;
    }
}

int main() {
    const int size = 1000;

    // Allocate host memory
    int *h_input = (int*)malloc(size * sizeof(int));
    int *h_output = (int*)malloc(size * sizeof(int));

    // Initialize host data
    for (int i = 0; i < size; i++) {
        h_input[i] = i;
        h_output[i] = 0;
    }

    // Allocate device memory
    int *d_input, *d_intermediate, *d_output;
    hipMalloc((void**)&d_input, size * sizeof(int));
    hipMalloc((void**)&d_intermediate, size * sizeof(int));
    hipMalloc((void**)&d_output, size * sizeof(int));

    //Create CUDA streams
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // Transfer data from host to device using stream1
    hipMemcpyAsync(d_input, h_input, size * sizeof(int), hipMemcpyHostToDevice, stream1);

    // Launch stage1Kernel in stream1
    dim3 blockDim(256);
    dim3 gridDim((size + blockDim.x - 1) / blockDim.x);

    // Launch stage1Kernel in stream1
    stage1Kernel<<<gridDim, blockDim, 0, stream1>>>(d_input, d_intermediate, size);

    // stream2가 stream1의 완료를 기다리도록 이벤트 사용
    hipEvent_t stage1_complete;
    hipEventCreate(&stage1_complete);
    hipEventRecord(stage1_complete, stream1);
    hipStreamWaitEvent(stream2, stage1_complete, 0);

    // Launch stage2Kernel in stream2 (stage1 완료 후)
    stage2Kernel<<<gridDim, blockDim, 0, stream2>>>(d_intermediate, d_output, size);

    // Transfer data from device to host using stream2
    hipMemcpyAsync(h_output, d_output, size * sizeof(int), hipMemcpyDeviceToHost, stream2);

    // Synchronize streams
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    // Cleanup event
    hipEventDestroy(stage1_complete);

    // Print results
    for (int i = 0; i < size; i++) {
       std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    // Cleanup
    hipFree(d_input);
    hipFree(d_intermediate);
    hipFree(d_output);
    free(h_input);
    free(h_output);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    
    return 0;
}